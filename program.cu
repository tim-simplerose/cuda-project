#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>

#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file,
                               const int line) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d) %s.\n",
            file, line, errorMessage, static_cast<int>(err),
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__global__ void testKernel(float *g_idata, float *g_odata) {
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float) num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

void run(int argc, char **argv) {
    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    hipMalloc((void **) &d_idata, mem_size);
    // copy host memory to device
    hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

    // allocate device memory for result
    float *d_odata;
    hipMalloc((void **) &d_odata, mem_size);

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads, hipMemcpyDeviceToHost);

    printf("Results:\n");
    for(unsigned int i = 0; i < num_threads; ++i)
        printf("%f\n", h_odata[i]);


    // cleanup memory
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);
}

int main(int argc, char **argv) {
    run(argc, argv);
}
